#include "hip/hip_runtime.h"
#include "solve.h"

#define H 8192
#define WO 8192
#define WI 8194

void solve(float *input, float *output) 
{
    float *dev_input = NULL;
    hipMalloc((void**)&dev_input, WI*H*sizeof(float));
    hipMemcpy(dev_input, input, WI*H*sizeof(float),hipMemcpyHostToDevice);
    int idx = 8190, j = 0;
    printf("origin=\n");
    for(int ki=0; ki<3; ++ki){
        for(int kj=0; kj<3; ++kj){
            printf("%.2lf  ",input[(idx+ki)*WI+j+kj]);
        }
        printf("\n");
    }

    float *dev_output = NULL;
    hipMalloc((void**)&dev_output, WO*H*sizeof(float));
    kernel<<<128,64>>>(dev_input, dev_output);

    hipMemcpy(output, dev_output, WO*H*sizeof(float),hipMemcpyDeviceToHost);


}

__global__ void kernel(float* input, float* output){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int j=0; j<WO; ++j){
        float tmp = 0;
        for(int ki=0; ki<3; ++ki){
            for(int kj=0; kj<3; ++kj){
                tmp += input[(idx+ki)*WI+j+kj];
            }
        }
        if(idx*WO+j == 67092480){
            for(int ki=0; ki<3; ++ki){
                for(int kj=0; kj<3; ++kj){
                    printf("%.2lf  ",input[(idx+ki)*WI+j+kj]);
                }
                printf("\n");
            }
            printf("idx = [%d,%d]=%d tmp = %.2lf\n", idx, j, idx*WO+j, tmp);
        }
        output[idx*WO+j] = tmp / 9;
    }
}